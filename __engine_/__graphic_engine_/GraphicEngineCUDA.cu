#include "hip/hip_runtime.h"

#include "GraphicEngineCUDA.cuh"
#include "__graphic_engine_/GraphicEngine.h"


void GraphicEngine::AllocateVertex2D() {

	unsigned int vertexs2d_size = data_info_.numberOfVertexs * sizeof(Vertex2D);

	hipMalloc((void**)& device_vertexs_2d_, vertexs2d_size);
	hipHostMalloc((void**)& host_vertexs_2d_, vertexs2d_size);

};


__global__ void ProjectVertexs(const Vertex3D* const vertexs_3d, Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const Vertex3D cameraPosition, const Vector3D vectorX, const Vector3D vectorY, const Vector3D vectorZ, const float k) {

		int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (threadIndex < number_of_vertexs) {

		//Offset

		float offset_x = vertexs_3d[threadIndex].x - cameraPosition.x;
		float offset_y = vertexs_3d[threadIndex].y - cameraPosition.y;
		float offset_z = vertexs_3d[threadIndex].z - cameraPosition.z;

		float new_x = offset_x * vectorX.x + offset_y * vectorX.y + offset_z * vectorX.z;
		float new_y = offset_x * vectorY.x + offset_y * vectorY.y + offset_z * vectorY.z;
		float new_z = offset_x * vectorZ.x + offset_y * vectorZ.y + offset_z * vectorZ.z;
		
		vertexs_2d[threadIndex].x = ( k * new_x / (new_z + k) ) ;
		vertexs_2d[threadIndex].y = ( k * new_y / (new_z + k) ) ;

	}

}

__global__ void ConvertInDisplayCoordinats(Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const unsigned int display_width, const unsigned int display_height) {
	
	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;

	if (threadIndex < number_of_vertexs) {

		vertexs_2d[threadIndex].x = vertexs_2d[threadIndex].x * 1000.0f + display_width / 2;
		vertexs_2d[threadIndex].y = display_height / 2 - vertexs_2d[threadIndex].y * 1000.0f;
		
	}

}
__global__ void DrawLines(const Vertex2D* const vertexs_2d, const Polygon3D* const device_polygons, const unsigned int number_of_polygons, RgbPixel* const display_buffer, const unsigned int display_width, const unsigned int display_height) {

	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;

	if (threadIndex < number_of_polygons * 3) {

		const unsigned int polygon_number = threadIndex / 3;
		const unsigned int vertex_number = threadIndex % 3;
	
		const unsigned int first_vertex_index = device_polygons[polygon_number].ratios[vertex_number].vertexNumber;
		const unsigned int second_vertex_number = ((vertex_number + 1) < 3) ? (vertex_number + 1) : 0;
		const unsigned int second_vertex_index = device_polygons[polygon_number].ratios[second_vertex_number].vertexNumber ;

		int x1 = vertexs_2d[first_vertex_index ].x;
		int y1 = vertexs_2d[first_vertex_index].y;
		const int x2 = vertexs_2d[second_vertex_index].x;
		const int y2 = vertexs_2d[second_vertex_index].y;

		const bool coordinats_are_correct = (x1 > 0 && x1 < display_width) && (x2 > 0 && x2 < display_width) && (y1 > 0 && y1 < display_height) && (y2 > 0 && y2 < display_height);

		if (coordinats_are_correct) {

			const int deltaX = abs(x2 - x1);
			const int deltaY = abs(y2 - y1);
			const int signX = x1 < x2 ? 1 : -1;
			const int signY = y1 < y2 ? 1 : -1;
			//
			int error = deltaX - deltaY;

			display_buffer[display_width * y2 + x2].rgb_reserved = 0;
			display_buffer[display_width * y2 + x2].rgb_red = 255;
			display_buffer[display_width * y2 + x2].rgb_green = 0;
			display_buffer[display_width * y2 + x2].rgb_blue = 0;

			while (x1 != x2 || y1 != y2)
			{

				display_buffer[display_width * y1 + x1].rgb_reserved = 0;
				display_buffer[display_width * y1 + x1].rgb_red = 255;
				display_buffer[display_width * y1 + x1].rgb_green = 0;
				display_buffer[display_width * y1 + x1].rgb_blue = 0;

				const int error2 = error * 2;
				//
				if (error2 > -deltaY)
				{
					error -= deltaY;
					x1 += signX;
				}

				if (error2 < deltaX)
				{
					error += deltaX;
					y1 += signY;
				}

			}
		}
	}

}
__global__ void  SetScreenColor(RgbPixel* const device_display_buffer, const RgbPixel rgb_pixel, const unsigned int number_of_pixels) {

	int thread_index = threadIdx.x + blockIdx.x * blockDim.x;

	if (thread_index < number_of_pixels) device_display_buffer[thread_index] = rgb_pixel;

}

void GraphicEngine::SetDisplayBufferColor(const RgbColor& rgb_color)
{
	const RgbPixel rgb_pixel = { rgb_color.rgb_blue, rgb_color.rgb_green, rgb_color.rgb_red, rgb_color.rgb_reserved };

	const unsigned int number_of_threads = 1024;

	const unsigned int number_of_blocks = (1920 * 1080 * sizeof(RgbPixel) + number_of_threads - 1) / number_of_threads;

	SetScreenColor <<< number_of_blocks, number_of_threads >>> (this->device_display_buffer_, rgb_pixel, 1920 * 1080);


}
void CPUCountingProjectVertexs(const Vertex3D* const vertexs_3d, Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const Vertex3D cameraPosition, const Vector3D vectorX, const Vector3D vectorY, const Vector3D vectorZ, const float k) {

	

	for (size_t i = 0; i < number_of_vertexs; i++)
	{



		//Offset
		const float offset_x = vertexs_3d[i].x - cameraPosition.x;
		const float offset_y = vertexs_3d[i].y - cameraPosition.y;
		const float offset_z = vertexs_3d[i].z - cameraPosition.z;

		const float new_x = offset_x * vectorX.x + offset_y * vectorX.y + offset_z * vectorX.z;
		const float new_y = offset_x * vectorY.x + offset_y * vectorY.y + offset_z * vectorY.z;
		const float new_z = offset_x * vectorZ.x + offset_y * vectorZ.y + offset_z * vectorZ.z;

		vertexs_2d[i].x = (k * new_x / (new_z + k));
		vertexs_2d[i].y = (k * new_y / (new_z + k));
	}
	
}
void CPUCountingConvertInDisplayCoordinats(Vertex2D* const vertexs_2d, const unsigned int number_of_vertexs, const unsigned int display_width, const unsigned int display_height) {

	

	for (size_t i = 0; i < number_of_vertexs; i++)
	{

		vertexs_2d[i].x = vertexs_2d[i].x * (float) 1000 + display_width / 2;
		vertexs_2d[i].y = display_height / 2 - vertexs_2d[i].y * 1000;

	}
}
void CPUCountingDrawLines(const Vertex2D* const vertexs_2d, const Polygon3D* const device_polygons, const unsigned int number_of_polygons, RgbPixel* const display_buffer, const unsigned int display_width, const unsigned int display_height) {

	{
		const unsigned int number_of_lines = number_of_polygons * 3;

		for (size_t i = 0; i < number_of_lines; i++)
		{

			const unsigned int polygon_number = i / 3;
			const unsigned int vertex_number = i % 3;

			const unsigned int first_vertex_index = device_polygons[polygon_number].ratios[vertex_number].vertexNumber;
			const unsigned int second_vertex_number = ((vertex_number + 1) < 3) ? (vertex_number + 1) : 0;
			const unsigned int second_vertex_index = device_polygons[polygon_number].ratios[second_vertex_number].vertexNumber;


			int x1 = vertexs_2d[first_vertex_index].x;
			int y1 = vertexs_2d[first_vertex_index].y;
			int x2 = vertexs_2d[second_vertex_index].x;
			int y2 = vertexs_2d[second_vertex_index].y;

			const bool coordinats_are_correct = (x1 > 0 && x1 < display_width) && (x2 > 0 && x2 < display_width) && (y1 > 0 && y1 < display_height) && (y2 > 0 && y2 < display_height);

			if (coordinats_are_correct) {

				const int deltaX = abs(x2 - x1);
				const int deltaY = abs(y2 - y1);
				const int signX = x1 < x2 ? 1 : -1;
				const int signY = y1 < y2 ? 1 : -1;
				//
				int error = deltaX - deltaY;

				display_buffer[display_width * y2 + x2].rgb_reserved = 0;
				display_buffer[display_width * y2 + x2].rgb_red = 255;
				display_buffer[display_width * y2 + x2].rgb_green = 0;
				display_buffer[display_width * y2 + x2].rgb_blue = 0;

				/*display_buffer[display_width * y1 + x1].rgb_reserved = 0;
				display_buffer[display_width * y1 + x1].rgb_red = 255;
				display_buffer[display_width * y1 + x1].rgb_green = 0;
				display_buffer[display_width * y1 + x1].rgb_blue = 0;*/


				while (x1 != x2 || y1 != y2)
				{
					display_buffer[display_width * y1 + x1].rgb_reserved = 0;
					display_buffer[display_width * y1 + x1].rgb_red = 255;
					display_buffer[display_width * y1 + x1].rgb_green = 0;
					display_buffer[display_width * y1 + x1].rgb_blue = 0;

					const int error2 = error * 2;
					//
					if (error2 > -deltaY)
					{
						error -= deltaY;
						x1 += signX;
					}
					if (error2 < deltaX)
					{
						error += deltaX;
						y1 += signY;
					}
				}

			}
		}
	}
}
__global__ void draw(RgbPixel* display_buffer) {
	int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

	display_buffer[thread_id].rgb_green = 255;

}

CameraInfo GraphicEngine::GetCameraInfo() {
	CameraInfo info;
	info.camera_pos = (*camera_->GetPosition());
	info.dis_proj_plane = camera_->GetDistanceToProjPlane();
	info.vector_x = camera_->GetVectorX();
	info.vector_y = camera_->GetVectorY();
	info.vector_z = camera_->GetDirection();
	return info;
}
void GraphicEngine::TestFunction()
{


}

void GraphicEngine::CreateMeshFrame() {

	const Vertex3D const  camera_position  = *(camera_->GetPosition());
	
	const Vector3D vector_x = camera_->GetVectorX();
	const Vector3D vector_y = camera_->GetVectorY();
	const Vector3D vector_z = camera_->GetDirection();

	const float distance_to_projection_plane = camera_->GetDistanceToProjPlane();

	const Vertex3D* const device_vertexs_3d = device_data_.deviceVertexs;
	Vertex2D* const device_vertexs_2d = device_vertexs_2d_;
	const Polygon3D* const device_polygons = device_data_.devicePolygons;

	
	RgbColor color;
	color.rgb_blue = 20;
	color.rgb_green = 255;
	color.rgb_red = 0;

	const unsigned int number_of_threads = 1024;
	unsigned int number_of_blocks = (data_info_.numberOfVertexs + number_of_threads - 1) / number_of_threads;

	SetDisplayBufferColor(color);

	ProjectVertexs <<<number_of_blocks, number_of_threads>>> (device_vertexs_3d, device_vertexs_2d, data_info_.numberOfVertexs, camera_position, vector_x, vector_y, vector_z, distance_to_projection_plane);

	ConvertInDisplayCoordinats <<<number_of_blocks, number_of_threads >>> (device_vertexs_2d, data_info_.numberOfVertexs, display_width_, display_height_);

	number_of_blocks = (data_info_.numberOfPolygons * 3 + number_of_threads - 1) / number_of_threads;

	DrawLines <<<number_of_blocks, number_of_threads >>> (device_vertexs_2d, device_polygons, data_info_.numberOfPolygons, device_display_buffer_, display_width_, display_height_);	

	//hipMemcpy((void**)host_display_buffer_, device_display_buffer_, display_buffer_size_, hipMemcpyDeviceToHost);
	//const Vertex3D* const host_vertexs_3d = data_info_.allVertexs;
	//hipMemcpy(host_display_buffer_, device_display_buffer_, size_of_display_buffer_, hipMemcpyDeviceToHost);
	//CPUCountingProjectVertexs(host_vertexs_3d, this->host_vertexs_2d_, this->data_info_.numberOfVertexs, camera_position, vector_x, vector_y, vector_z, distance_to_projection_plane);
	//CPUCountingConvertInDisplayCoordinats(host_vertexs_2d_, this->data_info_.numberOfVertexs, display_width_, display_height_);
	//Polygon3D* host_polygons = this->data_info_.allPolygons;
	//CPUCountingDrawLines(host_vertexs_2d_, host_polygons, this->data_info_.numberOfPolygons, this->host_display_buffer_, display_width_, display_height_);
	//hipMemcpy(host_display_buffer_, device_display_buffer_, size_of_display_buffer_, hipMemcpyDeviceToHost);

}

__global__ void DrawPolygons(z_element* z_buffer, Vertex2D* vertexs_2d, Polygon3D* polygons, Vertex3D* vertexs_3d) {

	int thread_index = threadIdx.x + blockDim.x * blockIdx.x;

	Polygon3D polygon = polygons[thread_index];

	Vertex2D f_vertex_2d = vertexs_2d[polygon.ratios[0].vertexNumber];
	float f_z = vertexs_3d[polygon.ratios[0].vertexNumber].z;

	Vertex2D s_vertex_2d = vertexs_2d[polygon.ratios[1].vertexNumber];
	float s_z = vertexs_3d[polygon.ratios[1].vertexNumber].z;

	Vertex2D t_vertex_2d = vertexs_2d[polygon.ratios[2].vertexNumber];
	float t_z = vertexs_3d[polygon.ratios[2].vertexNumber].z;



}

void GraphicEngine::CreateFlatFrame() {

	const Vertex3D const  camera_position = *(camera_->GetPosition());

	const Vector3D vector_x = camera_->GetVectorX();
	const Vector3D vector_y = camera_->GetVectorY();
	const Vector3D vector_z = camera_->GetDirection();

	const float distance_to_projection_plane = camera_->GetDistanceToProjPlane();

	const Vertex3D* const device_vertexs_3d = device_data_.deviceVertexs;
	Vertex2D* const device_vertexs_2d = device_vertexs_2d_;
	const Polygon3D* const device_polygons = device_data_.devicePolygons;

	RgbColor color;
	color.rgb_blue = 20;
	color.rgb_green = 255;
	color.rgb_red = 0;

	const unsigned int number_of_threads = 1024;
	unsigned int number_of_blocks = (data_info_.numberOfVertexs + number_of_threads - 1) / number_of_threads;

	SetDisplayBufferColor(color);

	ProjectVertexs <<<number_of_blocks, number_of_threads >>> (device_vertexs_3d, device_vertexs_2d, data_info_.numberOfVertexs, camera_position, vector_x, vector_y, vector_z, distance_to_projection_plane);

	ConvertInDisplayCoordinats <<<number_of_blocks, number_of_threads >>> (device_vertexs_2d, data_info_.numberOfVertexs, display_width_, display_height_);

	number_of_blocks = (data_info_.numberOfPolygons * 3 + number_of_threads - 1) / number_of_threads;

	DrawPolygons <<<number_of_blocks, number_of_threads >>> ();

}



